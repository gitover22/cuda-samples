#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This example shows how to use the clock function to measure the performance
 * of block of threads of a kernel accurately. Blocks are executed in parallel
 * and out of order. Since there's no synchronization mechanism between blocks,
 * we measure the clock once for each block. The clock samples are written to
 * device memory.
 */

// System includes
#include <assert.h>
#include <stdint.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>



/**
 * @brief 执行定时的归约操作，计算输入数组的最小值并记录每个块的执行时间。
 * 
 * 这个 CUDA 内核函数从输入数组中执行并行归约操作以找到最小值，并使用 `clock()` 函数记录每个块的执行时间。
 * 
 * @param input  输入数组，存储每个线程块要处理的数据。
 * @param output 输出数组，存储每个块计算的最小值。
 * @param timer  定时器数组，记录每个块的开始和结束时间。
 */
__global__ static void timedReduction(const float *input, float *output,
                                      clock_t *timer) {
  // __shared__ float shared[2 * blockDim.x];
  extern __shared__ float shared[]; // 分配一个动态大小的共享内存

  const int tid = threadIdx.x; // （0 - 255） 获取当前线程的索引
  const int bid = blockIdx.x;// （0 - 63）  获取当前块的索引

  if (tid == 0) timer[bid] = clock(); // 块的第一个线程开始计时

  // 将全局内存中的输入数据拷贝到共享内存中
  shared[tid] = input[tid];
  shared[tid + blockDim.x] = input[tid + blockDim.x];

  // 开始执行归约操作以找到最小值
  for (int d = blockDim.x; d > 0; d /= 2) {
    // 同步线程，确保所有线程都完成了共享内存的写入操作
    __syncthreads();
    // 只有前一半的线程参与本轮比较
    if (tid < d) {
      float f0 = shared[tid];
      float f1 = shared[tid + d];
      // 如果后面的元素比前面的元素小，则更新为较小值
      if (f1 < f0) {
        shared[tid] = f1;
      }
    }
  }

  // 当归约完成后，第 0 号线程将最小值写入输出数组
  if (tid == 0) output[bid] = shared[0];
  // 再次同步所有线程，确保归约操作完成
  __syncthreads();
  // 如果当前线程是第 0 号线程，记录当前块的结束时钟周期
  if (tid == 0) timer[bid + gridDim.x] = clock();
}

#define NUM_BLOCKS 64
#define NUM_THREADS 256

int main(int argc, char **argv) {
  printf("CUDA Clock sample\n");

  int dev = findCudaDevice(argc, (const char **)argv);

  float *dinput = NULL;
  float *doutput = NULL;
  clock_t *dtimer = NULL;

  clock_t timer[NUM_BLOCKS * 2];
  float input[NUM_THREADS * 2];

  for (int i = 0; i < NUM_THREADS * 2; i++) {
    input[i] = (float)i;
  }

  checkCudaErrors(hipMalloc((void **)&dinput, sizeof(float) * NUM_THREADS * 2));
  checkCudaErrors(hipMalloc((void **)&doutput, sizeof(float) * NUM_BLOCKS));
  checkCudaErrors(hipMalloc((void **)&dtimer, sizeof(clock_t) * NUM_BLOCKS * 2));

  checkCudaErrors(hipMemcpy(dinput, input, sizeof(float) * NUM_THREADS * 2,
                             hipMemcpyHostToDevice));

  timedReduction<<<NUM_BLOCKS, NUM_THREADS, sizeof(float) * 2 * NUM_THREADS>>>(dinput, doutput, dtimer);

  checkCudaErrors(hipMemcpy(timer, dtimer, sizeof(clock_t) * NUM_BLOCKS * 2, hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(dinput));
  checkCudaErrors(hipFree(doutput));
  checkCudaErrors(hipFree(dtimer));

  long double avgElapsedClocks = 0;

  for (int i = 0; i < NUM_BLOCKS; i++) {
    avgElapsedClocks += (long double)(timer[i + NUM_BLOCKS] - timer[i]);
  }

  avgElapsedClocks = avgElapsedClocks / NUM_BLOCKS;
  printf("Average clocks/block = %Lf\n", avgElapsedClocks);

  return EXIT_SUCCESS;
}
